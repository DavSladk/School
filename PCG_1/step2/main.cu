#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2022)
 * Projekt c. 1 (cuda)
 * Login: xsladk07
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <utility>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Time measurement
  struct timeval t1, t2;

  if (argc != 10)
  {
    printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    exit(1);
  }

  // Number of particles
  const int N           = std::stoi(argv[1]);
  // Length of time step
  const float dt        = std::stof(argv[2]);
  // Number of steps
  const int steps       = std::stoi(argv[3]);
  // Number of thread blocks
  const int thr_blc     = std::stoi(argv[4]);
  // Write frequency
  int writeFreq         = std::stoi(argv[5]);
  // number of reduction threads
  const int red_thr     = std::stoi(argv[6]);
  // Number of reduction threads/blocks
  const int red_thr_blc = std::stoi(argv[7]);

  // Size of the simulation CUDA gird - number of blocks
  const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
  // Size of the reduction CUDA grid - number of blocks
  const size_t reductionGrid  = (red_thr + red_thr_blc - 1) / red_thr_blc;

  // Log benchmark setup
  printf("N: %d\n", N);
  printf("dt: %f\n", dt);
  printf("steps: %d\n", steps);
  printf("threads/block: %d\n", thr_blc);
  printf("blocks/grid: %lu\n", simulationGrid);
  printf("reduction threads/block: %d\n", red_thr_blc);
  printf("reduction blocks/grid: %lu\n", reductionGrid);

  const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  writeFreq = (writeFreq > 0) ?  writeFreq : 0;


  t_particles particles_cpu;

  // Vypocet size s padding tak, aby v kernelech nemusela byt podminka, zda vlakno jeste nevypadlo z rozsahu
  int size = 0;
  int totalThreads = simulationGrid * thr_blc;
  if( N == totalThreads )
  {
    size = N;
  }
  else if( N < totalThreads )
  {
    size = totalThreads;
  }
  else
  {
    if(N % totalThreads == 0)
    {
      size = N;
    }
    else
    {
      size = ( (N / totalThreads ) + 1 ) * totalThreads;
    }
  }

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                            FILL IN: CPU side memory allocation (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  particles_cpu.pos_x  = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  particles_cpu.pos_y  = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  particles_cpu.pos_z  = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  
  particles_cpu.vel_x  = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  particles_cpu.vel_y  = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  particles_cpu.vel_z  = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  
  particles_cpu.weight = (float *)aligned_alloc(ALIGMENT, size * sizeof(float));
  
  memset(particles_cpu.pos_x,  0, size * sizeof(float));
  memset(particles_cpu.pos_y,  0, size * sizeof(float));
  memset(particles_cpu.pos_z,  0, size * sizeof(float));
  
  memset(particles_cpu.vel_x,  0, size * sizeof(float));
  memset(particles_cpu.vel_y,  0, size * sizeof(float));
  memset(particles_cpu.vel_z,  0, size * sizeof(float));
  
  memset(particles_cpu.weight, 0, size * sizeof(float));

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                              FILL IN: memory layout descriptor (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                      Stride of two               Offset of the first
   *  Data pointer        consecutive elements        element in floats,
   *                      in floats, not bytes        not bytes
  */
  MemDesc md(
        particles_cpu.pos_x,                1,              0,              // Postition in X
        particles_cpu.pos_y,                1,              0,              // Postition in Y
        particles_cpu.pos_z,                1,              0,              // Postition in Z
        particles_cpu.vel_x,                1,              0,              // Velocity in X
        particles_cpu.vel_y,                1,              0,              // Velocity in Y
        particles_cpu.vel_z,                1,              0,              // Velocity in Z
        particles_cpu.weight,               1,              0,              // Weight
        N,                                                                  // Number of particles
        recordsNum);                                                        // Number of records in output file

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::cerr<<e.what()<<std::endl;
    return -1;
  }


  t_particles  particles_one_gpu;
  t_particles  particles_two_gpu;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                  FILL IN: GPU side memory allocation (step 0)                                    //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  hipMalloc<float>(&(particles_one_gpu.pos_x),      size * sizeof(float));
  hipMalloc<float>(&(particles_one_gpu.pos_y),      size * sizeof(float));
  hipMalloc<float>(&(particles_one_gpu.pos_z),      size * sizeof(float));  
  hipMalloc<float>(&(particles_one_gpu.vel_x),      size * sizeof(float));
  hipMalloc<float>(&(particles_one_gpu.vel_y),      size * sizeof(float));
  hipMalloc<float>(&(particles_one_gpu.vel_z),      size * sizeof(float));  
  hipMalloc<float>(&(particles_one_gpu.weight),     size * sizeof(float));

  hipMalloc<float>(&(particles_two_gpu.pos_x),      size * sizeof(float));
  hipMalloc<float>(&(particles_two_gpu.pos_y),      size * sizeof(float));
  hipMalloc<float>(&(particles_two_gpu.pos_z),      size * sizeof(float));  
  hipMalloc<float>(&(particles_two_gpu.vel_x),      size * sizeof(float));
  hipMalloc<float>(&(particles_two_gpu.vel_y),      size * sizeof(float));
  hipMalloc<float>(&(particles_two_gpu.vel_z),      size * sizeof(float));  
  hipMalloc<float>(&(particles_two_gpu.weight),     size * sizeof(float));


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                       FILL IN: memory transfers (step 0)                                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  hipMemcpy(particles_one_gpu.pos_x,  particles_cpu.pos_x,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_one_gpu.pos_y,  particles_cpu.pos_y,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_one_gpu.pos_z,  particles_cpu.pos_z,  size * sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(particles_one_gpu.vel_x,  particles_cpu.vel_x,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_one_gpu.vel_y,  particles_cpu.vel_y,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_one_gpu.vel_z,  particles_cpu.vel_z,  size * sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(particles_one_gpu.weight, particles_cpu.weight, size * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(particles_two_gpu.pos_x,  particles_cpu.pos_x,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_two_gpu.pos_y,  particles_cpu.pos_y,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_two_gpu.pos_z,  particles_cpu.pos_z,  size * sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(particles_two_gpu.vel_x,  particles_cpu.vel_x,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_two_gpu.vel_y,  particles_cpu.vel_y,  size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_two_gpu.vel_z,  particles_cpu.vel_z,  size * sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(particles_two_gpu.weight, particles_cpu.weight, size * sizeof(float), hipMemcpyHostToDevice);

  gettimeofday(&t1, 0);

  int sharedParticles = thr_blc;
  int sharedMemSize = 7 * sharedParticles * sizeof(float);

  for(int s = 0; s < steps; s++)
  {
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: kernels invocation (step 0)                                     //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    calculate_velocity<<<simulationGrid, thr_blc, sharedMemSize>>>(particles_one_gpu, particles_two_gpu, N, dt, sharedParticles);
    std::swap(particles_one_gpu, particles_two_gpu);


    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                          FILL IN: synchronization  (step 4)                                    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    if (writeFreq > 0 && (s % writeFreq == 0))
    {
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //                          FILL IN: synchronization and file access logic (step 4)                             //
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    }
  }


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  hipDeviceSynchronize();

  gettimeofday(&t2, 0);

  // Approximate simulation wall time
  double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
  printf("Time: %f s\n", t);
  printf("Time per step: %f s\n", t/steps);


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                             FILL IN: memory transfers for particle data (step 0)                                 //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnGPU;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnCPU = centerOfMassCPU(md);

  std::cout << "Center of mass on CPU:" << std::endl
            << comOnCPU.x <<", "
            << comOnCPU.y <<", "
            << comOnCPU.z <<", "
            << comOnCPU.w
            << std::endl;

  std::cout << "Center of mass on GPU:" << std::endl
            << comOnGPU.x<<", "
            << comOnGPU.y<<", "
            << comOnGPU.z<<", "
            << comOnGPU.w
            << std::endl;

  // Writing final values to the file
  h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
  h5Helper.writeParticleDataFinal();

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
