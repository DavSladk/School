#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2022)
 * Projekt c. 1 (cuda)
 * Login: xsladk07
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  float r, dx, dy, dz;
  float vx, vy, vz;
  float r3, G_dt_r3, Fg_dt_m2_r;

  float tmp_x = 0;
  float tmp_y = 0;
  float tmp_z = 0;

  float isComputable = 0;

  int index = blockDim.x * blockIdx.x + threadIdx.x;

  for(int i = 0; i < N; i++)
  {
    // (index != i) <- make sure, you are not computing effects of particle on itself
    // (r > COLLISION_DISTANCE) <- because it was in the cpu implementation
    isComputable = (index != i) && (r > COLLISION_DISTANCE);

    dx = p.pos_x[index] - p.pos_x[i];
    dy = p.pos_y[index] - p.pos_y[i];
    dz = p.pos_z[index] - p.pos_z[i];

    r = sqrt(dx*dx + dy*dy + dz*dz);

    r3 = r * r * r + FLT_MIN;
    G_dt_r3 = -G * dt / r3;
    Fg_dt_m2_r = G_dt_r3 * p.weight[i];

    vx = Fg_dt_m2_r * dx;
    vy = Fg_dt_m2_r * dy;
    vz = Fg_dt_m2_r * dz;

    tmp_x += vx * isComputable;
    tmp_y += vy * isComputable;
    tmp_z += vz * isComputable;
  }
  
  tmp_vel.vel_x[index] += tmp_x;
  tmp_vel.vel_y[index] += tmp_y;
  tmp_vel.vel_z[index] += tmp_z;
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate collision velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  float r, dx, dy, dz;
  float vx, vy, vz;

  float tmp_x = 0;
  float tmp_y = 0;
  float tmp_z = 0;

  float isComputable = 0;

  int index = blockDim.x * blockIdx.x + threadIdx.x;

  for(int i = 0; i < N; i++)
  {
    isComputable = (index != i) && (r < COLLISION_DISTANCE) && (r > 0.0f);

    dx = p.pos_x[index] - p.pos_x[i];
    dy = p.pos_y[index] - p.pos_y[i];
    dz = p.pos_z[index] - p.pos_z[i];

    r = sqrt(dx*dx + dy*dy + dz*dz);

    vx = ((p.weight[index] * p.vel_x[index] - p.weight[i] * p.vel_x[index] + 2 * p.weight[i] * p.vel_x[i]) / (p.weight[index] + p.weight[i])) - p.vel_x[index];
    vy = ((p.weight[index] * p.vel_y[index] - p.weight[i] * p.vel_y[index] + 2 * p.weight[i] * p.vel_y[i]) / (p.weight[index] + p.weight[i])) - p.vel_y[index];
    vz = ((p.weight[index] * p.vel_z[index] - p.weight[i] * p.vel_z[index] + 2 * p.weight[i] * p.vel_z[i]) / (p.weight[index] + p.weight[i])) - p.vel_z[index];

    tmp_x += vx * isComputable;
    tmp_y += vy * isComputable;
    tmp_z += vz * isComputable;
  }

  tmp_vel.vel_x[index] += tmp_x;
  tmp_vel.vel_y[index] += tmp_y;
  tmp_vel.vel_z[index] += tmp_z;

}// end of calculate_collision_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  p.vel_x[index] += tmp_vel.vel_x[index];
  p.vel_y[index] += tmp_vel.vel_y[index];
  p.vel_z[index] += tmp_vel.vel_z[index];

  p.pos_x[index] += p.vel_x[index] * dt;
  p.pos_y[index] += p.vel_y[index] * dt;
  p.pos_z[index] += p.vel_z[index] * dt;
}// end of update_particle
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p_in    - particles input
 * @param p_out   - particles output
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
  float r, dx, dy, dz;
  float vx, vy, vz;
  float r3, G_dt_r3, Fg_dt_m2_r;

  float tmp_x = 0;
  float tmp_y = 0;
  float tmp_z = 0;

  float isComputable = 0;

  int index = blockDim.x * blockIdx.x + threadIdx.x;

  for(int i = 0; i < N; i++)
  {
    // GRAVITY


    dx = p_in.pos_x[index] - p_in.pos_x[i];
    dy = p_in.pos_y[index] - p_in.pos_y[i];
    dz = p_in.pos_z[index] - p_in.pos_z[i];

    r = sqrt(dx*dx + dy*dy + dz*dz);

    // (index != i) <- make sure, you are not computing effects of particle on itself
    // (r > COLLISION_DISTANCE) <- because it was in the cpu implementation
    isComputable = (index != i) && (r > COLLISION_DISTANCE);

    r3 = r * r * r + FLT_MIN;
    G_dt_r3 = -G * dt / r3;
    Fg_dt_m2_r = G_dt_r3 * p_in.weight[i];

    vx = Fg_dt_m2_r * dx;
    vy = Fg_dt_m2_r * dy;
    vz = Fg_dt_m2_r * dz;

    tmp_x += vx * isComputable;
    tmp_y += vy * isComputable;
    tmp_z += vz * isComputable;

    // COLLISION

    isComputable = (index != i) && (r < COLLISION_DISTANCE) && (r > 0.0f);

    vx = ((p_in.weight[index] * p_in.vel_x[index] - p_in.weight[i] * p_in.vel_x[index] + 2 * p_in.weight[i] * p_in.vel_x[i]) / (p_in.weight[index] + p_in.weight[i])) - p_in.vel_x[index];
    vy = ((p_in.weight[index] * p_in.vel_y[index] - p_in.weight[i] * p_in.vel_y[index] + 2 * p_in.weight[i] * p_in.vel_y[i]) / (p_in.weight[index] + p_in.weight[i])) - p_in.vel_y[index];
    vz = ((p_in.weight[index] * p_in.vel_z[index] - p_in.weight[i] * p_in.vel_z[index] + 2 * p_in.weight[i] * p_in.vel_z[i]) / (p_in.weight[index] + p_in.weight[i])) - p_in.vel_z[index];

    tmp_x += vx * isComputable;
    tmp_y += vy * isComputable;
    tmp_z += vz * isComputable;
  }

  p_out.vel_x[index] += tmp_x;
  p_out.vel_y[index] += tmp_y;
  p_out.vel_z[index] += tmp_z;

  p_out.pos_x[index] += p_out.vel_x[index] * dt;
  p_out.pos_y[index] += p_out.vel_y[index] * dt;
  p_out.pos_z[index] += p_out.vel_z[index] * dt;

}

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
